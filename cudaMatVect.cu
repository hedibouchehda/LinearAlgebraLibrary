#include "hip/hip_runtime.h"
#include "SparseMatrix.h" 

#define threadsPerBlock 128 
#define blocksPerGrid 128 



__global__ void kernel(ColumnBegEndInLine* d_begEndLines, StructCOO* d_columnsValues, double* d_vect, double* d_result, int* d_N)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x; 

    int columnIndex(0);
    int nonZerosOnLine(0); 

    while (i<d_N[0])
    {
        d_result[i] = 0;
        
        for (int j=d_begEndLines[i].beg; j<=d_begEndLines[i].end; j++)
        {
            d_result[i] +=  d_columnsValues[j].value * d_vect[d_columnsValues[j].columnIndex];       
        }
               
        i += blockDim.x*gridDim.x;
    }
   
}



Vector cudaMatVect(SparseMatrix* spMat, Vector& vect)
{   
    
    spMat->setBegEndOnLine(); 
    
    Vector result(spMat->getNumLines()); 
   
    int* N = (int*) malloc(sizeof(int)); 
    N[0] = sp->getNumLines();
      
    
    
    size_t ColumnBegEndInLineSize = spMat->getNumLines()*sizeof(ColumnBegEndInLine); 
    size_t inputSize = vect.m_size*sizeof(int); 
    size_t outputSize = (spMat->getNumLines())*sizeof(int); 
    size_t StructCOOSize = spMat->getNumOfNonZeros()*sizeof(StructCOO);

    //allocating variables on the device 
    ColumnBegEndInLine* d_begEndLines;
    StructCOO* d_columnsValues;
    double* d_vect; 
    double* d_result; 
    int* d_N; 
    
    hipMalloc((void**)&d_begEndLines,ColumnBegEndInLineSize); 
    hipMalloc((void**)&result,outputSize); 
    hipMalloc((void**)&d_columnsValues,StructCOOSize); 
    hipMalloc((void**)&d_vect,inputSize); 
    hipMalloc((void**)&d_N,sizeof(int)); 

    //copyng to device
    hipMemcpy(d_vect,vect.vect,inputSize,hipMemcpyHostToDevice);
    hipMemcpy(d_N,N,sizeof(int),hipMemcpyHostToDevice); 
    hipMemcpy(d_begEndLines,spMat->m_begEndOnLine,ColumnBegEndInLineSize,hipMemcpyHostToDevice);
    hipMemcpy(d_columnsValues,spMat->m_columnValue,StructCOOSize,hipMemcpyHostToDevice); 

    kernel<<< blocksPerGrid,threadsPerBlock >>>(d_begEndLines,d_columnsValues,d_vect,d_result,d_N); 

    hipMemcpy(result.vect,d_result,outputSize,hipMemcpyDeviceToHost); 

    free(spMat->m_begEndOnLine);
    
    hipFree(d_begEndLines);
    hipFree(result);
    hipFree(d_columnsValues);
    hipFree(d_vect);
    hipFree(d_N);
     
    return result;
}

int main()
{
    SparseMatrix* sp = new SparseMatrix(100,100); 
    std::vector<double> vect; 
    
    int i; 

    for (i=0; i<100; i++)
    {
        sp->add(i,i,1);
        vect.push_back(1.); 
    }

    sp->set(); 

    std::vector<double> result; 

    //result = cudaMatVect(sp,vect); 
    double* c_vect; 
    int* linesCOO;
    StructCOO* columnsValues;
    int* N;
    convertToCTypes(sp, vect, c_vect, linesCOO, columnsValues); 
    bool structArrayIsTrue(true), c_vectIsTrue(true), linesCOOIsTrue(true); 
      
    for (i=0; i<vect.size(); i++)
    {
        if (vect[i] != c_vect[i])
            c_vectIsTrue = false; 
    }
/*
    for (i=0; i<sp->m_linesCOO.size(); i++ )
    {
        if (sp->m_linesCOO[i]) 
            linesCOOIsTrue = false; 
    }

    for (i=0; i<sp->m_columnValue.size(); i++)
    {
        if (sp->m_columnValue[i] == columnsValues[i])
            structArrayIsTrue = false;           
    }
*/
    std::cout<<structArrayIsTrue<<" "<<linesCOOIsTrue<<" "<<c_vectIsTrue<<std::endl;
    return 0;
}